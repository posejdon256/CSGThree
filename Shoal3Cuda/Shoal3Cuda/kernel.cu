#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <string>
#include <cstdio>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>

#pragma comment(lib, "glew32.lib")
#include <iostream>
#include <ctime>
#include <Windows.h>
#include<>


using namespace std;

#define LEN 40


float arrayOfFishes[5 * LEN * LEN];

using namespace std;

__host__ __device__
float getAngle(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return 180 - atan2(-y1, x1) * 57.0;
}

__host__ __device__
float vectorMultiply(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return value;
}

__host__ __device__
float getVectorLength(float x1, float y1, float x2, float y2) {
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}

void defineFishes() {
	int numberOfFishesInRow = LEN;
	for (int i = 0; i < numberOfFishesInRow; i++) {
		for (int j = 0; j < numberOfFishesInRow * 5; j += 5) {
			arrayOfFishes[i * numberOfFishesInRow * 5 + j] = i * numberOfFishesInRow * 5 + j;
			arrayOfFishes[i * numberOfFishesInRow * 5 + j + 1] = -1.0f;
			arrayOfFishes[i * numberOfFishesInRow * 5 + j + 2] = 0.0f;
			arrayOfFishes[i * numberOfFishesInRow * 5 + j + 3] = 0.9f + i * -0.1f;
			arrayOfFishes[i * numberOfFishesInRow * 5 + j + 4] = 0.9f + j * -0.1f;
		}
	}
}
float* getArrayOfFishes() {
	return arrayOfFishes;
}
__host__
void updateInNeighborhoud(int x, int y) {
	float nei = 0.2;
	float neiClose = 0.1;
	int friends[LEN * LEN];
	int toClose[LEN * LEN];
	int friendsInd = 0;
	int toCloseInd = 0;
	float currentId = arrayOfFishes[x * LEN * 5 + y];
	float currentDirX = arrayOfFishes[x * LEN * 5 + y + 1];
	float currentDirY = arrayOfFishes[x * LEN * 5 + y + 2];
	float currentX = arrayOfFishes[x * LEN * 5 + y + 3];
	float currentY = arrayOfFishes[x * LEN * 5 + y + 4];

	for (int i = 0; i < LEN; i++) {
		for (int j = 0; j < 5 * LEN; j += 5) {
			float sFishId = arrayOfFishes[i * LEN * 5 + j];
			float sFishDirX = arrayOfFishes[i * LEN * 5 + j + 1];
			float sFishDirY = arrayOfFishes[i * LEN * 5 + j + 2];
			float sFishX = arrayOfFishes[i * LEN * 5 + j + 3];
			float sFishY = arrayOfFishes[i * LEN * 5 + j + 4];

			if (getVectorLength(sFishX, sFishY, currentX, currentY) < nei
				&& currentId != sFishId
				&& vectorMultiply(sFishX - currentX, sFishY - currentY, currentDirX, currentDirY) > 0) {
				friends[friendsInd++] = (i * LEN * 5 + j);
			}
			if (getVectorLength(sFishX, sFishY, currentX, currentY) < neiClose
				&& currentId != sFishId
				&& vectorMultiply(sFishX - currentX, sFishY - currentY, currentDirX, currentDirY) > 0) {
				toClose[toCloseInd++] = (i * LEN * 5 + j);
			}
		}
	}
	float dirXNew = 0.0;
	float posNewX = 0.0;
	float posNewY = 0.0;
	float dirYNew = 0.0;
	float awayFromX = 0.0;
	float awayFromY = 0.0;
	for (int i = 0; i < friendsInd; i++) {
		dirXNew += arrayOfFishes[friends[i] + 1];
		dirYNew += arrayOfFishes[friends[i] + 2];

		posNewX += arrayOfFishes[friends[i] + 3];
		posNewY += arrayOfFishes[friends[i] + 4];

	}
	for (int i = 0; i < toCloseInd; i++) {
		awayFromX += currentX - arrayOfFishes[friends[i] + 3];
		awayFromY += currentY - arrayOfFishes[friends[i] + 4];
	}
	if (friendsInd == 0) {
		return;
	}
	if (toCloseInd == 0) {
		currentDirX += (dirXNew / (float)friendsInd) * 0.1;
		currentDirY += (dirYNew / (float)friendsInd) * 0.1;
		currentDirX += (posNewX / (float)friendsInd) * 0.05;
		currentDirY += (posNewY / (float)friendsInd) * 0.05;
	}
	else {
		currentDirX += (awayFromX / (float)friendsInd) * 5;
		currentDirY += (awayFromY / (float)friendsInd) * 5;
	}
	float vecLen = sqrt(pow(currentDirX, 2) + pow(currentDirY, 2));
	currentDirX /= vecLen;
	currentDirY /= vecLen;
	/*if (toClose.size() != 0) {
		current.x += current.dirX * 0.005;
		current.y += current.dirY * 0.005;
	}*/
	arrayOfFishes[x * LEN * 5 + y] = currentId;
	arrayOfFishes[x * LEN * 5 + y + 1] = currentDirX;
	arrayOfFishes[x * LEN * 5 + y + 2] = currentDirY;
	arrayOfFishes[x * LEN * 5 + y + 3] = currentX;
	arrayOfFishes[x * LEN * 5 + y + 4] = currentY;

}

__device__
void updateInNeighborhoudGpu(float *d_arrayOfFishes, int ind) {
	float nei = 0.2;
	float neiClose = 0.1;
	int friends[LEN * LEN];
	int toClose[LEN * LEN];
	int friendsInd = 0;
	int toCloseInd = 0;
	float currentId = d_arrayOfFishes[ind];
	float currentDirX = d_arrayOfFishes[ind + 1];
	float currentDirY = d_arrayOfFishes[ind + 2];
	float currentX = d_arrayOfFishes[ind + 3];
	float currentY = d_arrayOfFishes[ind + 4];

	for (int i = 0; i < LEN; i++) {
		for (int j = 0; j < 5 * LEN; j += 5) {
			float sFishId = d_arrayOfFishes[i * LEN * 5 + j];
			float sFishDirX = d_arrayOfFishes[i * LEN * 5 + j + 1];
			float sFishDirY = d_arrayOfFishes[i * LEN * 5 + j + 2];
			float sFishX = d_arrayOfFishes[i * LEN * 5 + j + 3];
			float sFishY = d_arrayOfFishes[i * LEN * 5 + j + 4];

			if (getVectorLength(sFishX, sFishY, currentX, currentY) < nei
				&& currentId != sFishId
				&& vectorMultiply(sFishX - currentX, sFishY - currentY, currentDirX, currentDirY) > 0) {
				friends[friendsInd++] = (i * LEN * 5 + j);
			}
			if (getVectorLength(sFishX, sFishY, currentX, currentY) < neiClose
				&& currentId != sFishId
				&& vectorMultiply(sFishX - currentX, sFishY - currentY, currentDirX, currentDirY) > 0) {
				toClose[toCloseInd++] = (i * LEN * 5 + j);
			}
		}
	}
	float dirXNew = 0.0;
	float posNewX = 0.0;
	float posNewY = 0.0;
	float dirYNew = 0.0;
	float awayFromX = 0.0;
	float awayFromY = 0.0;
	for (int i = 0; i < friendsInd; i++) {
		dirXNew += d_arrayOfFishes[friends[i] + 1];
		dirYNew += d_arrayOfFishes[friends[i] + 2];

		posNewX += d_arrayOfFishes[friends[i] + 3];
		posNewY += d_arrayOfFishes[friends[i] + 4];

	}
	for (int i = 0; i < toCloseInd; i++) {
		awayFromX += currentX - d_arrayOfFishes[friends[i] + 3];
		awayFromY += currentY - d_arrayOfFishes[friends[i] + 4];
	}
	if (friendsInd == 0) {
		return;
	}
	if (toCloseInd == 0) {
		currentDirX += (dirXNew / (float)friendsInd) * 0.1;
		currentDirY += (dirYNew / (float)friendsInd) * 0.1;
		currentDirX += (posNewX / (float)friendsInd) * 0.05;
		currentDirY += (posNewY / (float)friendsInd) * 0.05;
	}
	else {
		currentDirX += (awayFromX / (float)friendsInd) * 5;
		currentDirY += (awayFromY / (float)friendsInd) * 5;
	}
	float vecLen = sqrt(pow(currentDirX, 2) + pow(currentDirY, 2));
	currentDirX /= vecLen;
	currentDirY /= vecLen;
	/*if (toClose.size() != 0) {
		current.x += current.dirX * 0.005;
		current.y += current.dirY * 0.005;
	}*/
	d_arrayOfFishes[ind] = currentId;
	d_arrayOfFishes[ind + 1] = currentDirX;
	d_arrayOfFishes[ind + 2] = currentDirY;
	d_arrayOfFishes[ind + 3] = currentX;
	d_arrayOfFishes[ind + 4] = currentY;

}

__global__
void updateShoalGpu(float *d_arrayOfFishes ) {
	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = threadID; i < LEN * LEN * 5; i += numThreads + 5) {
		float sFishId = d_arrayOfFishes[i];
		float sFishDirX = d_arrayOfFishes[i + 1];
		float sFishDirY = d_arrayOfFishes[i + 2];
		float sFishX = d_arrayOfFishes[i + 3];
		float sFishY = d_arrayOfFishes[i + 4];


		if (sFishX <= -0.9 && sFishDirX < 0) {
			sFishDirX = -sFishDirX;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
			//sFishdirY = 1.0;
		}
		if (sFishX >= 0.9 && sFishDirX > 0) {
			sFishDirX = -sFishDirX;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
			//sFishdirY = -1.0;
		}
		if (sFishY <= -0.9 && sFishDirY < 0) {
			//sFishdirX = -1.0;
			sFishDirY = -sFishDirY;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
		}
		if (sFishY >= 0.9 && sFishDirY > 0) {
			//sFishdirX = 1.0;
			sFishDirY = -sFishDirY;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
		}
		sFishX += sFishDirX * 0.005;
		sFishY += sFishDirY * 0.005;

		d_arrayOfFishes[i] = sFishId;
		d_arrayOfFishes[i + 1] = sFishDirX;
		d_arrayOfFishes[i + 2] = sFishDirY;
		d_arrayOfFishes[i + 3] = sFishX;
		d_arrayOfFishes[i + 4] = sFishY;

	}
	for (int i = threadID; i < LEN * LEN * 5; i += numThreads + 5) 
	{
		updateInNeighborhoudGpu(d_arrayOfFishes, i);
	}
}


void updateShoal() {
	for (int i = 0; i < LEN * LEN * 5; i += 5) {
		float sFishId = arrayOfFishes[i];
		float sFishDirX = arrayOfFishes[i + 1];
		float sFishDirY = arrayOfFishes[i + 2];
		float sFishX = arrayOfFishes[i + 3];
		float sFishY = arrayOfFishes[i + 4];


		if (sFishX <= -0.9 && sFishDirX < 0) {
			sFishDirX = -sFishDirX;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
			//sFishdirY = 1.0;
		}
		if (sFishX >= 0.9 && sFishDirX > 0) {
			sFishDirX = -sFishDirX;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
			//sFishdirY = -1.0;
		}
		if (sFishY <= -0.9 && sFishDirY < 0) {
			//sFishdirX = -1.0;
			sFishDirY = -sFishDirY;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
		}
		if (sFishY >= 0.9 && sFishDirY > 0) {
			//sFishdirX = 1.0;
			sFishDirY = -sFishDirY;
			sFishX += sFishDirX * 0.01;
			sFishY += sFishDirY * 0.01;
		}
		sFishX += sFishDirX * 0.005;
		sFishY += sFishDirY * 0.005;

		arrayOfFishes[i] = sFishId;
		arrayOfFishes[i + 1] = sFishDirX;
		arrayOfFishes[i + 2] = sFishDirY;
		arrayOfFishes[i + 3] = sFishX;
		arrayOfFishes[i + 4] = sFishY;

	}
	for (int i = 0; i < LEN; i++) {
		for (int j = 0; j < 5 * LEN; j += 5) {
			int place = i * LEN * 5 + j;
			updateInNeighborhoud(i, j);
		}
	}
}

void changeViewPort(int w, int h)
{
	glViewport(0, 0, w, h);
}

void renderTriangle() {

	glColor3f(255.0 / 255.0, 204.0 / 255.0, 0.0 / 255.0);

	glVertex3f(-0.75, 0.5, 0.0);
	glVertex3f(1.0, 0.0, 0.0);
	glVertex3f(1.0, 1.0, 0.0);

}

void renderShoal() {
	float scale = 0.015;
	int  numberOfFishesInRow = LEN;
	float* fishes = getArrayOfFishes();
	for (int i = 0; i < LEN * LEN * 5; i += 5) {

		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();
		glTranslatef(fishes[i + 3], fishes[i + 4], 0.0);
		float angle = getAngle(fishes[i + 1], fishes[i + 2], 1, 0);
		glRotatef(angle, 0, 0, 1);
		glScalef(scale, scale, scale);

		glBegin(GL_POLYGON);
		renderTriangle();
		glEnd();
	}
	//glFlush();
}//
void renderGpu()
{
	float *d_fishes;
	hipMalloc((void**)&d_fishes, 5 * LEN * LEN * sizeof(float));
	hipMemcpy(d_fishes, arrayOfFishes, LEN * LEN * 5 * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	updateShoalGpu << <1024, 1024 >> > (d_fishes);
	hipDeviceSynchronize();
	hipMemcpy(arrayOfFishes, d_fishes, LEN * LEN * 5 * sizeof(float), hipMemcpyDeviceToHost);

}

void render()
{	//while(true) {
	glClearColor(64.0 / 255.0, 164.0 / 255.0, 223.0 / 225.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	//const clock_t begin_time = clock();
	//while (clock() - begin_time < 10);
	
	renderGpu();
	//updateShoal();
	renderShoal();
	glutSwapBuffers();
	glutPostRedisplay();
	//}
}



int main(int argc, char* argv[])
{
	// Initialize GLUTx
	glutInit(&argc, argv);
	// Set up some memory buffers for our display
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	// Set the window size
	glutInitWindowSize(1000, 800);
	// Create the window with the title "Hello,GL"
	glutCreateWindow("Shoal");
	defineFishes();
	// Bind the two functions (above) to respond when necessary
	glutReshapeFunc(changeViewPort);
	glutDisplayFunc(render);
	glutMainLoop();
	//glutMainLoop();

	// Very important!  This initializes the entry points in the OpenGL driver so we can 
	// call all the functions in the API.
	GLenum err = glewInit();
	if (GLEW_OK != err) {
		fprintf(stderr, "GLEW error");
		return 1;
	}

	return 0;
}