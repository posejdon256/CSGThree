#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <string>
#include <cstdio>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>
#include <utility>  

#pragma comment(lib, "glew32.lib")
#include <iostream>
#include <ctime>
#include <Windows.h>
#include<>
#include <algorithm>

using namespace std;

#define LEN 1000


struct Sphere
{
	int r;
	int positionX;
	int positionY;
};
enum Operation
{
	Sum = 0,
	Mul = 1,
	Diff = 2,
	None = 3
};
struct node {
	Operation operation;
	Sphere* sphere;
	node* left = NULL;
	node* right = NULL;
	node* parent = NULL;
};
struct zLen {
	bool isIn;
	float pos;
};
node * root;
unsigned char *data;

__host__ __device__
float getAngle(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return 180 - atan2(-y1, x1) * 57.0;
}

__host__ __device__
float vectorMultiply(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return value;
}

__host__ __device__
float getVectorLength(float x1, float y1, float x2, float y2) {
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}

__global__
void updateShoalGpu(float *d_arrayOfFishes) {
	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadID;
}
vector<zLen*>* prepareZetsArray() {
	vector<zLen*> *zlenth = new vector<zLen*>[LEN * LEN];
	for (int i = 0; i < LEN * LEN; i++) {
		zlenth = new vector<zLen*>();
	}
	return zlenth;
}
bool sortFunction(zLen* i, zLen* j) { return i->pos > j->pos; }
unsigned char* prepareData() {
	unsigned char *data = new unsigned char[LEN * LEN * 4];
	for (int i = 0; i < LEN * LEN * 4; i++) {
		data[i] = 0;
	}

	return data;
}
void changeViewPort(int w, int h)
{
	glViewport(0, 0, w, h);
}

void renderTriangle() {

	glColor3f(255.0 / 255.0, 204.0 / 255.0, 0.0 / 255.0);

	glVertex3f(-0.75, 0.5, 0.0);
	glVertex3f(1.0, 0.0, 0.0);
	glVertex3f(1.0, 1.0, 0.0);

}

void renderShoal() {
}
void renderGpu()
{
}
float getZet(int x, int y, int cenX, int cenY, int r) {
	return sqrt(pow(r, 2) - pow(x - cenX, 2) - pow(y - cenY, 2));
}
float zetDist(int x, int y, int cenX, int cenY, int r) {
	return  max(((200.0f - getZet(x, y, cenX, cenY, r)) / 500.0f) * 255.0f, 0.0f);
}
float zetDistBack(int x, int y, int cenX, int cenY, int r) {
	return  min(((200.0f + getZet(x, y, cenX, cenY, r)) / 500.0f) * 255.0f, 255.0f);
}
bool isInCircle(int x, int y, int cenX, int cenY, int r) {
	float len = sqrt(pow(x - cenX, 2) + pow(y - cenY, 2));
	return len <= r;
}
void setInd(unsigned char *data, int i, int j, int value) {
	data[((i + 500) * LEN + (j + 500)) * 4] = value;
	data[((i + 500) * LEN + (j + 500)) * 4 + 1] = value;
	data[((i + 500) * LEN + (j + 500)) * 4 + 2] = value;
	data[((i + 500) * LEN + (j + 500)) * 4 + 3] = value;
}
float getMinimumOfArray(vector<float> arr) {
	if (arr.size() == 0) {
		return 0;
	}
	float minimum = INFINITY;
	for (int i = 0; i < arr.size(); i++) {
		minimum = minimum < arr[i] ? minimum : arr[i];
	}
	return minimum;
}
bool sortByMulValue(pair<int, float> first, pair<int, float> second) {
	return first.second < second.second;
}
float getMultiplyValue(vector<float> arr) {
	if (arr.size() < 4) {
		return 0;
	}
	vector<pair<int, float>> _arr;
	for (int i = 0; i < 4; i++) {
		pair<int, float> elem(floor(i / 2), arr[i]);
		_arr.push_back(elem);
	}
	sort(_arr.begin(), _arr.end(), sortByMulValue);
	if (_arr[0].first != _arr[1].first) {
		return _arr[1].second;
	}
	return 0;
}
float getDifferenceValue(vector<float> arr, int id) {
	if (arr.size() < 2) {
		return 0;
	}
	vector<pair<int, float>> _arr;
	for (int i = 0; i < arr.size(); i++) {
		pair<int, float> elem(floor(i / 2), arr[i]);
		_arr.push_back(elem);
	}
	sort(_arr.begin(), _arr.end(), sortByMulValue);
	if (_arr.size() == 2 && id == 0) {
		return _arr[0].second;
	}
	else if (id == 1 && _arr.size() == 2) {
		return 0;
	}
	if (_arr[0].first == 0) {
		return _arr[0].second;
	}
	return _arr[2].second;
}
int getMinimum(Sphere* left, Sphere* right, bool X) {
	if (left == NULL || right == NULL) {
		return 0;
	}
	return X ? min(left->positionX - left->r, right->positionX - right->r) : min(left->positionY - left->r, right->positionY - right->r);
}
int getMaximum(Sphere* left, Sphere* right, bool X) {
	if (left == NULL || right == NULL) {
		return 0;
	}
	return X ? max(left->positionX + left->r, right->positionX + right->r) : max(left->positionY + left->r, right->positionY + right->r);
}
bool isCloseEnough(Sphere* sphere, int i, int j) {
	return sqrt(pow(sphere->positionX - i, 2) + pow(sphere->positionY - j, 2)) < sphere->r;
}
void DrawElement(node* Node, unsigned char *data, vector<zLen*>* zlength) {
	if (Node->operation == None) return;
	if (Node->operation != None) {
		DrawElement(Node->left, data, zlength);
	}
	if (Node->operation != None) {
		DrawElement(Node->right, data, zlength);
	}
	Sphere* left = Node->left->sphere;
	Sphere* right = Node->right->sphere;


	int minimumX = getMinimum(left, right, true);
	int maximumX = getMaximum(left, right, true);
	int minimumY = getMinimum(left, right, false);
	int maximumY = getMaximum(left, right, false);
	for (int i = minimumX; i < maximumX; i++) {
		for (int j = minimumY; j < maximumY; j++) {
			vector<float> distances;
			int _dist = 0;
			if (left != NULL && isCloseEnough(left, i, j)) {
				distances.push_back(zetDist(i, j, left->positionX, left->positionY, left->r));
				distances.push_back(zetDistBack(i, j, left->positionX, left->positionY, left->r));
			}
			if (right != NULL && isCloseEnough(right, i, j)) {
				_dist = 1;
				distances.push_back(zetDist(i, j, right->positionX, right->positionY, right->r));
				distances.push_back(zetDistBack(i, j, right->positionX, right->positionY, right->r));
			}
			if (Node->operation == Sum) {
				setInd(data, i, j, getMinimumOfArray(distances));
			}
			else if (Node->operation == Mul) {
				setInd(data, i, j, getMultiplyValue(distances));
			}
			else { // Diff
				setInd(data, i, j, getDifferenceValue(distances, _dist));
			}
		}
	}
	glDrawPixels(LEN, LEN, GL_RGBA, GL_UNSIGNED_BYTE, data);
}
void CreateRoot() {
	root = new node();
	root->operation = Sum;

	Sphere* sphere1 = new Sphere();
	sphere1->r = 200;
	sphere1->positionX = 100;
	sphere1->positionY = 100;

	Sphere* sphere3 = new Sphere();
	sphere3->r = 100;
	sphere3->positionX = -50;
	sphere3->positionY = -50;

	Sphere* sphere2 = new Sphere();
	sphere2->r = 100;
	sphere2->positionX = -120;
	sphere2->positionY = -120;

	Sphere* sphere4 = new Sphere();
	sphere4->r = 50;
	sphere4->positionX = 50;
	sphere4->positionY = -50;

	node* left1 = new node();
	left1->operation = Diff;
	left1->parent = root;

	node* right2 = new node();
	right2->operation = None;
	right2->sphere = sphere2;
	right2->parent = left1;

	node* left2 = new node();
	left2->operation = Sum;
	left2->parent = left1;

	left1->right = right2;
	left1->left = left2;

	node* right3 = new node();
	right3->operation = None;
	right3->sphere = sphere4;
	right3->parent = left2;

	node* left3 = new node();
	left3->operation = None;
	left3->sphere = sphere3;
	left3->parent = left2;

	left2->left = left3;
	left2->right = right3;

	node* right = new node();
	right->sphere = sphere1;
	right->operation = None;
	right->parent = root;
	root->right = right;
	root->left = left1;
}
void render()
{
	glClearColor(0.0 / 255.0, 0.0 / 255.0, 0.0 / 225.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	DrawElement(root, prepareData(), prepareZetsArray());


	renderGpu();
	renderShoal();
	glutSwapBuffers();
	//glutPostRedisplay();
}
int main(int argc, char* argv[])
{
	// Initialize GLUTx
	glutInit(&argc, argv);
	// Set up some memory buffers for our display
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	// Set the window size
	glutInitWindowSize(LEN, LEN);
	// Create the window with the title "Hello,GL"
	glutCreateWindow("CSGThree");
	CreateRoot();
	glutDisplayFunc(render);
	glutMainLoop();

	GLenum err = glewInit();
	if (GLEW_OK != err) {
		fprintf(stderr, "GLEW error");
		return 1;
	}

	return 0;
}