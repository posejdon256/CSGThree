#include "hip/hip_runtime.h"

#pragma region Includes

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <string>
#include <cstdio>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>
#include <utility>  

#pragma comment(lib, "glew32.lib")
#include <iostream>
#include <ctime>
#include <Windows.h>
#include<>
#include <algorithm>
#pragma endregion
using namespace std;
#define LEN 500

unsigned char ddata[LEN * LEN * 4];
#pragma region Structures
struct Position {
	float x;
	float y;
	float z;
};
struct Color {
	int r;
	int g;
	int b;
	int a;
};
struct Sphere
{
	int r;
	float position[3];
	int color[4];
};
enum Operation
{
	Sum = 0,
	Mul = 1,
	Diff = 2,
	None = 3
};
struct Line {
	Sphere* in;
	float inPosition;
	Sphere* out;
	float outPosition;

};
struct Node {
	Operation operation;
	Sphere* sphere;
	Node* left = NULL;
	Node* right = NULL;
	Node* parent = NULL;
	Line* lines;
};
Node* root;
Sphere** spheres = new Sphere*[4];
Node** nodes = new Node*[7];
#pragma endregion
#pragma region CUDA

__host__ __device__
float getAngle(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return 180 - atan2(-y1, x1) * 57.0;
}
__host__ __device__
float vectorMultiply(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return value;
}
__host__ __device__
float getVectorLength(float x1, float y1, float x2, float y2) {
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}
#pragma endregion
#pragma region  Helpers
Position* crossMultiply(Position* v1, Position* v2) {
	// [ a2 * b3 - a3 * b2, a3 * b1 - a1 * b3, a1 * b2 - a2 * b1 ]
	Position* newP = new Position();
	newP->x = v1->y * v2->z - v1->z * v2->y;
	newP->y = v1->z * v2->x - v1->x * v2->z;
	newP->z = v1->x * v2->y - v1->y * v2->x;
	return newP;
}
__device__ __host__
float dotMultiply(Position v1, Position v2) {
	return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}
__device__ __host__
Position DiffPoints(Position p1, Position p2) {
	Position p3 = Position();
	p3.x = p1.x - p2.x;
	p3.y = p1.y - p2.y;
	p3.z = p1.z - p2.z;
	return p3;
}
__device__ __host__
Position DividePoint(Position p, float a) {
	Position newP = Position();
	newP.x = p.x / a;
	newP.y = p.y / a;
	newP.z = p.z / a;
	return newP;
}
__device__ __host__
Position SumPoints(Position p1, Position p2) {
	Position newP = Position();
	newP.x = p1.x + p2.x;
	newP.y = p1.y + p2.y;
	newP.z = p1.z + p2.z;
	return newP;
}
__device__ __host__
Position MulPoints(Position p, float a) {
	Position newP = Position();
	newP.x = p.x * a;
	newP.y = p.y * a;
	newP.z = p.z *a;
	return newP;
}
float lengthBetweenTwoPoints(Position* p1, Position* p2) {
	return sqrt(pow(p1->x - p2->x, 2) + pow(p1->x - p2->y, 2) + pow(p1->z - p2->z, 2));
}
__device__ __host__
float vectorLength(Position p) {
	return sqrt(p.x * p.x + p.y * p.y + p.z * p.z);
}
#pragma endregion
#pragma region Logic operations
vector<Line*> MulOperation(vector<Line*> lines1, vector<Line*> lines2) {
	vector<Line*> ret;
	return ret;
}
vector<Line*> SumOperation(vector<Line*> lines1, vector<Line*> lines2) {
	vector<Line*> ret;
	Line* last = lines1[lines1.size() - 1] < lines2[lines2.size() - 1] ? lines2[lines2.size() - 1] : lines1[lines1.size() - 1];
	while (ret.size() == 0 || ret[ret.size() - 1] != last) {

	}
	return ret;
}
vector<Line*> DiffOperation(vector<Line*> lines, vector<Line*> line3) {
	vector<Line*> ret;
	return ret;
}
#pragma endregion
#pragma region Engine

__device__
void testUpdate(int i, unsigned char* d_data, int value, int* color) {
	d_data[i * 4] = color[0] * value;
	d_data[i * 4 + 1] = color[1] * value;
	d_data[i * 4 + 2] = color[2] * value;
	d_data[i * 4 + 3] = 255;
}
__host__ __device__
Line countLine(Sphere* sphere, Position* camera, int x, int y, int z) {
	Position place = Position();
	place.x = x;
	place.y = y;
	place.z = z;

	Position spherePosition = Position();
	spherePosition.x = sphere->position[0];
	spherePosition.y = sphere->position[1];
	spherePosition.z = sphere->position[2];

	Position v = DividePoint(DiffPoints(place, *camera), vectorLength(DiffPoints(place, *camera)));
	Position dv = DiffPoints(*camera, spherePosition);
	float a = dotMultiply(v, v);
	float b = 2 * dotMultiply(v, dv);
	float c = dotMultiply(dv, dv) - sphere->r * sphere->r;

	Line ret = Line();
	ret.in = sphere;
	ret.out = sphere;
	//Range result = { false, 0,0, sphere_id };

	float delta = b * b - 4 * a*c;

	if (delta < 0) {
		ret.inPosition = 0;
		return ret;
	}
	else {
		if (delta == 0)
		{
			float t = -b / (2 * a);
			ret.inPosition = t;
			ret.outPosition = t;
		}
		else
		{
			float sdelta = sqrt(delta);
			float t1 = (-b + sdelta) / (2 * a);
			float t2 = (-b - sdelta) / (2 * a);
			if (t1 < t2)
			{
				ret.inPosition = t1;
				ret.outPosition = t2;
			}
			else
			{
				ret.inPosition = t2;
				ret.outPosition = t1;
			}
		}
	}
	return ret;

}
__host__ __device__
void concatTwoNodes(Node* ret, int i) {
	Node* left = ret->left;
	Node* right = ret->right;
	if (left->lines[i * 8].inPosition != 0) {
		ret->lines[i * 8] = left->lines[i * 8];
		//ret->lines[i * 8].inPosition = 255;
	}
	else {
		ret->lines[i * 8] = right->lines[i * 8];
		//ret->lines[i * 8].inPosition = 255;
	}
}
__device__
void DrawElement(Node* node, Position* camera, unsigned char *d_data, int i) {
	if (node->left != NULL) {
		DrawElement(node->left, camera, d_data, i);
	}
	if (node->right != NULL) {
		DrawElement(node->right, camera, d_data, i);
	}
	if (node->sphere == NULL) {
		concatTwoNodes(node, i);
	}
	else {
		int y = i;
		while (y >= LEN) y -= LEN;
		Line sphereLine = countLine(node->sphere, camera, (int)(i / LEN), i % LEN, 0);
		node->lines[i * 8] = sphereLine;
	}
	if (node->parent == NULL) {
		testUpdate(i, d_data, (int)node->lines[i * 8].inPosition < 0 ? 0 : (int)node->lines[i * 8].inPosition, node->lines[i * 8].in->color);
	}
}
#pragma endregion
#pragma region THREE

__host__
Sphere* setSpherePosition(Sphere* sphere, int x, int y, int z) {
	sphere->position[0] = x;
	sphere->position[1] = y;
	sphere->position[2] = z;
	return sphere;
}
__host__
Sphere* setSphereColor(Sphere* sphere, int r, int g, int b, int a) {
	sphere->color[0] = r;
	sphere->color[1] = g;
	sphere->color[2] = b;
	sphere->color[3] = a;
	return sphere;
}
__host__
void CreateRoot() {
	hipMallocManaged((void **)&root, sizeof(Node*));
	//new Line*[LEN * LEN]
	hipMallocManaged((void **)&root->lines, sizeof(Line) * LEN * LEN * 8);
	root->parent = NULL;
	root->operation = Sum;

	Sphere* sphere1;
	hipMallocManaged((void **)&sphere1, sizeof(Sphere*));
	sphere1->r = 50;
	sphere1 = setSpherePosition(sphere1, 100, 100, 0);
	sphere1 = setSphereColor(sphere1, 255, 0, 255, 255);

	Sphere* sphere3;
	hipMallocManaged((void **)&sphere3, sizeof(Sphere*));
	sphere3->r = 70;
	sphere3 = setSpherePosition(sphere3, 150, 150, 0);
	sphere3 = setSphereColor(sphere3, 0, 255, 0, 255);

	Sphere* sphere2;
	hipMallocManaged((void **)&sphere2, sizeof(Sphere*));
	sphere2->r = 100;
	sphere2 = setSpherePosition(sphere2, 170, 140, 0);
	sphere2 = setSphereColor(sphere2, 0, 0, 255, 255);

	Sphere* sphere4;
	hipMallocManaged((void **)&sphere4, sizeof(Sphere*));
	sphere4->r = 50;
	sphere4 = setSpherePosition(sphere4, 600, 600, 0);
	sphere4 = setSphereColor(sphere4, 255, 0, 0, 255);

	Node* left1;
	hipMallocManaged((void **)&left1, sizeof(Node*));
	hipMallocManaged((void **)&left1->lines, sizeof(Line) * LEN * LEN * 8);
	left1->operation = Diff;
	left1->parent = root;

	Node* right2;
	hipMallocManaged((void **)&right2, sizeof(Node*));
	hipMallocManaged((void **)&right2->lines, sizeof(Line) * LEN * LEN * 8);
	right2->operation = None;
	right2->sphere = sphere2;
	right2->parent = left1;

	Node* left2;
	hipMallocManaged((void **)&left2, sizeof(Node*));
	hipMallocManaged((void **)&left2->lines, sizeof(Line) * 8 * LEN * LEN);
	left2->operation = Sum;
	left2->parent = left1;

	left1->right = right2;
	left1->left = left2;

	Node* right3;
	hipMallocManaged((void **)&right3, sizeof(Node*));
	hipMallocManaged((void **)&right3->lines, sizeof(Line) * 8 * LEN * LEN);
	right3->operation = None;
	right3->sphere = sphere4;
	right3->parent = left2;

	Node* left3;
	hipMallocManaged((void **)&left3, sizeof(Node*));
	hipMallocManaged((void **)&left3->lines, sizeof(Line)* 8 * LEN * LEN);
	left3->operation = None;
	left3->sphere = sphere3;
	left3->parent = left2;

	left2->left = left3;
	left2->right = right3;

	Node* right;
	hipMallocManaged((void **)&right, sizeof(Node*));
	hipMallocManaged((void **)&right->lines, sizeof(Line) * 8 * LEN * LEN);
	right->sphere = sphere1;
	right->operation = None;
	right->parent = root;

	root->right = right;
	root->left = left1;
}
#pragma endregion
#pragma region Render
__global__
void drawElements(unsigned char *d_data, Node* root, Position* camera) {

	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = (blockIdx.x * blockDim.x + threadIdx.x);

	int i = threadID % (LEN * LEN);
	DrawElement(root, camera, d_data, i);

}
void renderGpu()
{
	unsigned char *d_data;

	Position* camera;
	hipMallocManaged((void**)&camera, sizeof(Position*));
	camera->x = 0;
	camera->y = 0;
	camera->z = 1000;

	hipMalloc((void**)&d_data, LEN * LEN * 4 * sizeof(unsigned char));
	hipDeviceSynchronize();
	drawElements << < 1024, 1024 >> > (d_data, root, camera);
	hipDeviceSynchronize();
	hipMemcpy(ddata, d_data, LEN * LEN * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(d_data);

	glDrawPixels(LEN, LEN, GL_RGBA, GL_UNSIGNED_BYTE, ddata);

}
void render()
{
	glClearColor(0.0 / 255.0, 0.0 / 255.0, 0.0 / 225.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	renderGpu();
	//renderCpu();
	glutSwapBuffers();
	glutPostRedisplay();

}
#pragma endregion
int main(int argc, char* argv[])
{
	// Initialize GLUTx
	glutInit(&argc, argv);
	// Set up some memory buffers for our display
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	// Set the window size
	glutInitWindowSize(LEN, LEN);
	// Create the window with the title "Hello,GL"
	glutCreateWindow("CSGThree");
	CreateRoot();
	glutDisplayFunc(render);
	glutMainLoop();

	GLenum err = glewInit();
	if (GLEW_OK != err) {
		fprintf(stderr, "GLEW error");
		return 1;
	}

	return 0;
}