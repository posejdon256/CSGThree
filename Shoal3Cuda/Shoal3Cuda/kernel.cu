#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <string>
#include <cstdio>
#include <vector>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>

#pragma comment(lib, "glew32.lib")
#include <iostream>
#include <ctime>
#include <Windows.h>
#include<>


using namespace std;

#define LEN 10

struct  fish
{
	int id;
	float dirX;
	float dirY;
	float x;
	float y;
};

fish arrayOfFishes[LEN * LEN];

using namespace std;

float getAngle(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return 180 - atan2(-y1, x1) * 57.0;
}
float vectorMultiply(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return value;
}
float getVectorLength(float x1, float y1, float x2, float y2) {
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}

void defineFishes() {
	int numberOfFishesInRow = LEN;
	for (int i = 0; i < numberOfFishesInRow; i++) {
		for (int j = 0; j < numberOfFishesInRow; j++) {
			fish f;
			f.id = i * numberOfFishesInRow + j;
			f.dirX = -1.0;
			f.dirY = 0.0;
			f.x = 0.9 + i * -0.1;
			f.y = 0.9 + j * -0.1;

			arrayOfFishes[i * numberOfFishesInRow + j] = f;
		}
	}
}
fish* getArrayOfFishes() {
	return arrayOfFishes;
}

void updateInNeighborhoud(int x, int y) {
	float nei = 0.2;
	float neiClose = 0.1;
	vector<fish> friends;
	vector<fish> toClose;
	fish current = arrayOfFishes[x * LEN + y];
	for (int i = 0; i < LEN; i++) {
		for (int j = 0; j < LEN; j++) {
			fish sFish = arrayOfFishes[i * LEN + j];
			if (getVectorLength(sFish.x, sFish.y, current.x, current.y) < nei
				&& current.id != sFish.id
				&& vectorMultiply(sFish.x - current.x, sFish.y - current.y, current.dirX, current.dirY) > 0) {
				friends.push_back(sFish);
			}
			if (getVectorLength(sFish.x, sFish.y, current.x, current.y) < neiClose
				&& current.id != sFish.id
				&& vectorMultiply(sFish.x - current.x, sFish.y - current.y, current.dirX, current.dirY) > 0) {
				toClose.push_back(sFish);
			}
		}
	}
	float dirXNew = 0.0;
	float posNewX = 0.0;
	float posNewY = 0.0;
	float dirYNew = 0.0;
	float awayFromX = 0.0;
	float awayFromY = 0.0;
	for (int i = 0; i < friends.size(); i++) {
		dirXNew += friends[i].dirX;
		dirYNew += friends[i].dirY;

		posNewX += friends[i].x;
		posNewY += friends[i].y;

	}
	for (int i = 0; i < toClose.size(); i++) {
		awayFromX += (current.x - toClose[i].x);
		awayFromY += (current.y - toClose[i].y);
	}
	if (friends.size() == 0) {
		return;
	}
	if (toClose.size() == 0) {
		current.dirX += (dirXNew / (float)friends.size()) * 0.1;
		current.dirY += (dirYNew / (float)friends.size()) *0.1;
		current.dirX += (posNewX / (float)friends.size()) * 0.05;
		current.dirY += (posNewY / (float)friends.size()) * 0.05;
	}
	else {
		current.dirX += (awayFromX / (float)friends.size()) * 5;
		current.dirY += (awayFromY / (float)friends.size()) * 5;
	}
	float vecLen = sqrt(pow(current.dirX, 2) + pow(current.dirY, 2));
	current.dirX = current.dirX / vecLen;
	current.dirY = current.dirY / vecLen;
	/*if (toClose.size() != 0) {
		current.x += current.dirX * 0.005;
		current.y += current.dirY * 0.005;
	}*/
	arrayOfFishes[x * LEN + y] = current;
}

void updateShoal() {
	for (int i = 0; i < LEN * LEN; i++) {
		if (arrayOfFishes[i].x <= -0.9 && arrayOfFishes[i].dirX < 0) {
			arrayOfFishes[i].dirX = -arrayOfFishes[i].dirX;
			arrayOfFishes[i].x += arrayOfFishes[i].dirX * 0.01;
			arrayOfFishes[i].y += arrayOfFishes[i].dirY * 0.01;
			//arrayOfFishes[i].dirY = 1.0;
		}
		if (arrayOfFishes[i].x >= 0.9 && arrayOfFishes[i].dirX > 0) {
			arrayOfFishes[i].dirX = -arrayOfFishes[i].dirX;
			arrayOfFishes[i].x += arrayOfFishes[i].dirX * 0.01;
			arrayOfFishes[i].y += arrayOfFishes[i].dirY * 0.01;
			//arrayOfFishes[i].dirY = -1.0;
		}
		if (arrayOfFishes[i].y <= -0.9 && arrayOfFishes[i].dirY < 0) {
			//arrayOfFishes[i].dirX = -1.0;
			arrayOfFishes[i].dirY = -arrayOfFishes[i].dirY;
			arrayOfFishes[i].x += arrayOfFishes[i].dirX * 0.01;
			arrayOfFishes[i].y += arrayOfFishes[i].dirY * 0.01;
		}
		if (arrayOfFishes[i].y >= 0.9 && arrayOfFishes[i].dirY > 0) {
			//arrayOfFishes[i].dirX = 1.0;
			arrayOfFishes[i].dirY = -arrayOfFishes[i].dirY;
			arrayOfFishes[i].x += arrayOfFishes[i].dirX * 0.01;
			arrayOfFishes[i].y += arrayOfFishes[i].dirY * 0.01;
		}
		arrayOfFishes[i].x += arrayOfFishes[i].dirX * 0.005;
		arrayOfFishes[i].y += arrayOfFishes[i].dirY * 0.005;
	}
	for (int i = 0; i < LEN; i++) {
		for (int j = 0; j < LEN; j++) {
			int place = i * LEN + j;
			updateInNeighborhoud(i, j);
		}
	}
}

void changeViewPort(int w, int h)
{
	glViewport(0, 0, w, h);
}

void renderTriangle() {

	glColor3f(255.0 / 255.0, 204.0 / 255.0, 0.0 / 255.0);

	glVertex3f(-0.75, 0.5, 0.0);
	glVertex3f(1.0, 0.0, 0.0);
	glVertex3f(1.0, 1.0, 0.0);

}

void renderShoal() {
	float scale = 0.015;
	int  numberOfFishesInRow = 10;
	fish* fishes = getArrayOfFishes();
	for (int i = 0; i < 100; i++) {

		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();
		glTranslatef(fishes[i].x, fishes[i].y, 0.0);
		float angle = getAngle(fishes[i].dirX, fishes[i].dirY, 1, 0);
		glRotatef(angle, 0, 0, 1);
		glScalef(scale, scale, scale);

		glBegin(GL_POLYGON);
		renderTriangle();
		glEnd();
	}
	//glFlush();
}//

void render()
{	//while(true) {
	glClearColor(64.0 / 255.0, 164.0 / 255.0, 223.0 / 225.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	//const clock_t begin_time = clock();
	//while (clock() - begin_time < 10);

	updateShoal();
	renderShoal();
	glutSwapBuffers();
	glutPostRedisplay();
	//}
}



int main(int argc, char* argv[])
{
	// Initialize GLUTx
	glutInit(&argc, argv);
	// Set up some memory buffers for our display
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	// Set the window size
	glutInitWindowSize(1000, 800);
	// Create the window with the title "Hello,GL"
	glutCreateWindow("Shoal");
	defineFishes();
	// Bind the two functions (above) to respond when necessary
	glutReshapeFunc(changeViewPort);
	glutDisplayFunc(render);
	glutMainLoop();
	//glutMainLoop();

	// Very important!  This initializes the entry points in the OpenGL driver so we can 
	// call all the functions in the API.
	GLenum err = glewInit();
	if (GLEW_OK != err) {
		fprintf(stderr, "GLEW error");
		return 1;
	}

	return 0;
}