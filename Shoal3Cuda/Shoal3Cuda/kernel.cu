#include "hip/hip_runtime.h"

#pragma region Includes

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include <string>
#include <cstdio>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>
#include <utility>  

#pragma comment(lib, "glew32.lib")
#include <iostream>
#include <ctime>
#include <Windows.h>
#include<>
#include <algorithm>
#pragma endregion
using namespace std;
using namespace thrust;
#define LEN 1000


#pragma region Structures
struct Position {
	float x;
	float y;
	float z;
};
struct Color {
	int r;
	int g;
	int b;
	int a;
};
struct Sphere
{
	int r;
	Position position;
	Color color;
};
enum Operation
{
	Sum = 0,
	Mul = 1,
	Diff = 2,
	None = 3
};
struct Line {
	Sphere* in;
	float inPosition;
	Sphere* out;
	float outPosition;

};
struct Node {
	Operation operation;
	Sphere* sphere;
	Node* left = NULL;
	Node* right = NULL;
	Node* parent = NULL;
	thrust::device_vector<Line*> lines[LEN * LEN];
};
unsigned char *data;
#pragma endregion
#pragma region CUDA

__host__ __device__
float getAngle(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return 180 - atan2(-y1, x1) * 57.0;
}
__host__ __device__
float vectorMultiply(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return value;
}
__host__ __device__
float getVectorLength(float x1, float y1, float x2, float y2) {
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}
#pragma endregion
#pragma region  Helpers
Position* crossMultiply(Position* v1, Position* v2) {
	// [ a2 * b3 - a3 * b2, a3 * b1 - a1 * b3, a1 * b2 - a2 * b1 ]
	Position* newP = new Position();
	newP->x = v1->y * v2->z - v1->z * v2->y;
	newP->y = v1->z * v2->x - v1->x * v2->z;
	newP->z = v1->x * v2->y - v1->y * v2->x;
	return newP;
}
float dotMultiply(Position* v1, Position* v2) {
	return v1->x * v2->x + v1->y * v2->y + v1->z * v2->z;
}
unsigned char* prepareData() {
	unsigned char *data = new unsigned char[LEN * LEN * 4];
	for (int i = 0; i < LEN * LEN * 4; i++) {
		data[i] = 0;
	}

	return data;
}
Position* DiffPoints(Position* p1, Position* p2) {
	Position* p3 = new Position();
	p3->x = p1->x - p2->x;
	p3->y = p1->y - p2->y;
	p3->z = p1->z - p2->z;
	return p3;
}
Position* DividePoint(Position* p, float a) {
	Position* newP = new Position();
	newP->x = p->x / a;
	newP->y = p->y / a;
	newP->z = p->z / a;
	return newP;
}
Position* SumPoints(Position* p1, Position* p2) {
	Position* newP = new Position();
	newP->x = p1->x + p2->x;
	newP->y = p1->y + p2->y;
	newP->z = p1->z + p2->z;
	return newP;
}
Position* MulPoints(Position* p, float a) {
	Position* newP = new Position();
	newP->x = p->x * a;
	newP->y = p->y * a;
	newP->z = p->z *a;
	return newP;
}
float lengthBetweenTwoPoints(Position* p1, Position* p2) {
	return sqrt(pow(p1->x - p2->x, 2) + pow(p1->x - p2->y, 2) + pow(p1->z - p2->z, 2));
}
float vectorLength(Position* p) {
	return sqrt(pow(p->x, 2) + pow(p->y, 2) + pow(p->z, 2));
}
__host__ __device__
void setInd(unsigned char *data, int i, int value) {
	data[i * 4] = value;
	data[i * 4 + 1] = value;
	data[i * 4 + 2] = value;
	data[i * 4 + 3] = 255;
}
#pragma endregion
#pragma region Logic operations
vector<Line*> MulOperation(vector<Line*> lines1, vector<Line*> lines2) {
	vector<Line*> ret;
	return ret;
}
vector<Line*> SumOperation(vector<Line*> lines1, vector<Line*> lines2) {
	vector<Line*> ret;
	Line* last = lines1[lines1.size() - 1] < lines2[lines2.size() - 1] ? lines2[lines2.size() - 1] : lines1[lines1.size() - 1];
	while (ret.size() == 0 || ret[ret.size() - 1] != last) {

	}
	return ret;
}
vector<Line*> DiffOperation(vector<Line*> lines, vector<Line*> line3) {
	vector<Line*> ret;
	return ret;
}
#pragma endregion
#pragma region Engine
__host__ __device__
Line* countLine(Sphere* sphere, Position* camera, Position* place) {
	Position* v = DiffPoints(place, camera);
	Position* dv = DiffPoints(camera, &sphere->position);
	float a = dotMultiply(v, v);
	float b = 2 * dotMultiply(v, dv);
	float c = dotMultiply(dv, dv) - pow(sphere->r, 2);

	Line* ret = new Line();
	ret->in = sphere;
	ret->out = sphere;
	//Range result = { false, 0,0, sphere_id };

	float delta = b * b - 4 * a*c;

	if (delta < 0)
		return NULL;
	else
	{
		if (delta == 0)
		{
			float t = -b / (2 * a);
			ret->inPosition = t;
			ret->outPosition = t;
		}
		else
		{
			float sdelta = sqrt(delta);
			float t1 = (-b + sdelta) / (2 * a);
			float t2 = (-b - sdelta) / (2 * a);
			if (t1 < t2)
			{
				ret->inPosition = t1;
				ret->outPosition = t2;
			}
			else
			{
				ret->inPosition = t2;
				ret->outPosition = t1;
			}
		}
	}
	ret->inPosition = ret->inPosition < 0 ? ret->inPosition * -100 : ret->inPosition * 100;
	ret->outPosition = ret->outPosition < 0 ? ret->outPosition * -100 : ret->outPosition * 100;
	return ret;

}
__host__ __device__
void concatTwoNodes(Node* ret) {
	Node* left = ret->left;
	Node* right = ret->right;

	for (int i = 0; i < LEN * LEN; i++) {
		ret->lines[i].insert(ret->lines[i].end(), right->lines[i].begin(), right->lines[i].end());
		ret->lines[i].insert(ret->lines[i].end(), left->lines[i].begin(), left->lines[i].end());
	}
}
__device__
void DrawElement(Node* node, Position* camera, unsigned char *data, int i) {
	if (node->left != NULL) {
		DrawElement(node->left, camera, data, i);
	}
	if (node->right != NULL) {
		DrawElement(node->right, camera, data, i);
	}
	if (node->sphere == NULL) {
		concatTwoNodes(node);
	} else {
		Position* place = new Position();
		place->x = i % LEN;
		place->y = i;
		while (place->y >= LEN) place->y -= LEN;
		place->z = 0;
		Line* sphereLine = countLine(node->sphere, camera, place);
		if (sphereLine != NULL) {
			node->lines[i].push_back(sphereLine);
		}
	}
	if (node->parent == NULL) {
		setInd(data, i, node->lines[i].size() > 0 ? 1 : 0);
		//glDrawPixels(LEN, LEN, GL_RGBA, GL_UNSIGNED_BYTE, data);
	}
}
#pragma endregion
#pragma region THREE

__device__
Sphere* setSpherePosition(Sphere* sphere, int x, int y, int z) {
	sphere->position.x = x;
	sphere->position.y = y;
	sphere->position.z = z;
	return sphere;
}
__device__
Sphere* setSphereColor(Sphere* sphere, int r, int g, int b, int a) {
	sphere->color.r = r;
	sphere->color.g = g;
	sphere->color.b = b;
	sphere->color.a = a;
	return sphere;
}
__device__
Node* CreateRoot() {
	Node* root = new Node();
	root->parent = NULL;
	root->operation = Sum;

	Sphere* sphere1 = new Sphere();
	sphere1->r = 50;
	sphere1 = setSpherePosition(sphere1, 100, 100, 0);
	sphere1 = setSphereColor(sphere1, 255, 0, 255, 255);

	Sphere* sphere3 = new Sphere();
	sphere3->r = 70;
	sphere3 = setSpherePosition(sphere3, 150, 150, 0);
	sphere3 = setSphereColor(sphere3, 0, 255, 0, 255);

	Sphere* sphere2 = new Sphere();
	sphere2->r = 100;
	sphere2 = setSpherePosition(sphere2, 170, 140, 0);
	sphere2 = setSphereColor(sphere2, 0, 0, 255, 255);

	Sphere* sphere4 = new Sphere();
	sphere4->r = 120;
	sphere4 = setSpherePosition(sphere4, 200, 200, -50);
	sphere4 = setSphereColor(sphere4, 100, 100, 100, 255);

	Node* left1 = new Node();
	left1->operation = Diff;
	left1->parent = root;

	Node* right2 = new Node();
	right2->operation = None;
	right2->sphere = sphere2;
	right2->parent = left1;

	Node* left2 = new Node();
	left2->operation = Sum;
	left2->parent = left1;

	left1->right = right2;
	left1->left = left2;

	Node* right3 = new Node();
	right3->operation = None;
	right3->sphere = sphere4;
	right3->parent = left2;

	Node* left3 = new Node();
	left3->operation = None;
	left3->sphere = sphere3;
	left3->parent = left2;

	left2->left = left3;
	left2->right = right3;

	Node* right = new Node();
	right->sphere = sphere1;
	right->operation = None;
	right->parent = root;

	root->right = right;
	root->left = left1;
	return root;
}
#pragma endregion
#pragma region Render
void renderCpu() {
}
void initializeGPU() {
	//hipError_t error = hipSuccess;
	////size_t spheresSize = 4 * sizeof(Sphere);
	//size_t nodeSize = 8 * sizeof(Node);
	//size_t dataSize = 4 * LEN * LEN * sizeof(char);

	//error = hipMalloc((void**)&)
}
__global__
void DrawElements(Position* camera, unsigned char *data) {

	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;

	Node* root = CreateRoot();

	int i = threadID;
	if (threadID < LEN * LEN) {
		DrawElement(root, camera, data, i);
	}

}
void renderGpu()
{
	hipError_t error = hipSuccess;
	int pictureSize = LEN * LEN;
	size_t dataSize = 4 * pictureSize * sizeof(char);
	int threadsPerBlock = 256;

	Position* camera = new Position();
	camera->x = 0;
	camera->y = 0;
	camera->z = 1000;

	DrawElements << < 1024, 1024 >> > (camera, prepareData());
	
}
void render()
{
	glClearColor(0.0 / 255.0, 0.0 / 255.0, 0.0 / 225.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	//DrawElement(root, prepareData(), camera);


	renderGpu();
	//renderCpu();
	glutSwapBuffers();
	//glutPostRedisplay();
}
#pragma endregion
int main(int argc, char* argv[])
{
	// Initialize GLUTx
	glutInit(&argc, argv);
	// Set up some memory buffers for our display
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	// Set the window size
	glutInitWindowSize(LEN, LEN);
	// Create the window with the title "Hello,GL"
	glutCreateWindow("CSGThree");
	glutDisplayFunc(render);
	glutMainLoop();

	GLenum err = glewInit();
	if (GLEW_OK != err) {
		fprintf(stderr, "GLEW error");
		return 1;
	}

	return 0;
}