#include "hip/hip_runtime.h"

#pragma region Includes

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <string>
#include <cstdio>
#include <iterator>
#include <iostream>
#include <random>
#include <chrono>
#include <memory>
#include <functional>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include <gl/glew.h>
#include <gl/GL.h>
#include <gl/freeglut.h>
#include <utility>  

#pragma comment(lib, "glew32.lib")
#include <iostream>
#include <ctime>
#include <Windows.h>
#include<>
#include <algorithm>
#pragma endregion
using namespace std;
#define LEN 1000

#pragma region Structures
struct Position {
	float x;
	float y;
	float z;
};
struct Color {
	int r;
	int g;
	int b;
	int a;
};
struct Sphere
{
	int r;
	Position position;
	Color color;
};
enum Operation
{
	Sum = 0,
	Mul = 1,
	Diff = 2,
	None = 3
};
struct Line {
	Sphere* in;
	Position* inPosition;
	Sphere* out;
	Position* outPosition;

};
struct Node {
	Operation operation;
	Sphere* sphere;
	Node* left = NULL;
	Node* right = NULL;
	Node* parent = NULL;
	vector<Line*> lines;
};
Node * root;
unsigned char *data;
#pragma endregion
#pragma region CUDA

__host__ __device__
float getAngle(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return 180 - atan2(-y1, x1) * 57.0;
}
__host__ __device__
float vectorMultiply(float x1, float y1, float x2, float y2) {
	float value = x1 * x2 + y1 * y2;
	value = value != 0 ? value / sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2)) : 0;
	return value;
}
__host__ __device__
float getVectorLength(float x1, float y1, float x2, float y2) {
	return sqrt(pow(x1 - x2, 2) + pow(y1 - y2, 2));
}
#pragma endregion
#pragma region  Helpers
Position* crossMultiply(Position* v1, Position* v2) {
	// [ a2 * b3 - a3 * b2, a3 * b1 - a1 * b3, a1 * b2 - a2 * b1 ]
	Position* newP = new Position();
	newP->x = v1->y * v2->z - v1->z * v2->y;
	newP->y = v1->z * v2->x - v1->x * v2->z;
	newP->z = v1->x * v2->y - v1->y * v2->x;
	return newP;
}
float dotMultiply(Position* v1, Position* v2) {
	return v1->x * v2->x + v1->y * v2->y + v1->z * v2->z;
}
unsigned char* prepareData() {
	unsigned char *data = new unsigned char[LEN * LEN * 4];
	for (int i = 0; i < LEN * LEN * 4; i++) {
		data[i] = 0;
	}

	return data;
}
Position* DiffPoints(Position* p1, Position* p2) {
	Position* p3 = new Position();
	p3->x = p1->x - p2->x;
	p3->y = p1->y - p2->y;
	p3->z = p1->z - p2->z;
	return p3;
}
Position* DividePoint(Position* p, float a) {
	Position* newP = new Position();
	newP->x = p->x / a;
	newP->y = p->y / a;
	newP->z = p->z / a;
	return newP;
}
Position* SumPoints(Position* p1, Position* p2) {
	Position* newP = new Position();
	newP->x = p1->x + p2->x;
	newP->y = p1->y + p2->y;
	newP->z = p1->z + p2->z;
	return newP;
}
Position* MulPoints(Position* p, float a) {
	Position* newP = new Position();
	newP->x = p->x * a;
	newP->y = p->y * a;
	newP->z = p->z *a;
	return newP;
}
float lengthBetweenTwoPoints(Position* p1, Position* p2) {
	return sqrt(pow(p1->x - p2->x, 2) + pow(p1->x - p2->y, 2) + pow(p1->z - p2->z, 2));
}
float vectorLength(Position* p) {
	return sqrt(pow(p->x, 2) + pow(p->y, 2) + pow(p->z, 2));
}
void setInd(unsigned char *data, int i, int j, int value) {
	data[((i + 500) * LEN + (j + 500)) * 4] = value;
	data[((i + 500) * LEN + (j + 500)) * 4 + 1] = value;
	data[((i + 500) * LEN + (j + 500)) * 4 + 2] = value;
	data[((i + 500) * LEN + (j + 500)) * 4 + 3] = value;
}
#pragma endregion
#pragma region Logic operations
vector<Line*> MulOperation(vector<Line*> lines1, vector<Line*> lines2) {
}
vector<Line*> SumOperation(vector<Line*> lines1, vector<Line*> lines2) {
	vector<Line*> ret;
	Line* last = lines1[lines1.size() - 1] < lines2[lines2.size() - 1] ? lines2[lines2.size() - 1] : lines1[lines1.size() - 1];
	while (ret.size() == 0 || ret[ret.size() - 1] != last) {

	}
}
vector<Line*> DiffOperation(vector<Line*> lines, vector<Line*> line3) {

}
#pragma endregion
#pragma region Engine
Line* countLine(Sphere* sphere, Position* camera) {
	Position* l = DiffPoints(&sphere->position, camera);
	Line* ret = new Line();
	l = DividePoint(l, vectorLength(l));
	Position* cameraToCenter = DiffPoints(camera, &sphere->position);
	float a = dotMultiply(l, l);
	float b = 2 * dotMultiply(l, cameraToCenter);
	float c = dotMultiply(cameraToCenter, cameraToCenter) - pow(sphere->r, 2);
	if (pow(b, 2) < 4 * a * c) {
		return NULL;
	}
	float d1 = (-b + sqrt(pow(b, 2) - 4 * a * c)) / (2 * a);
	float d2 = (-b - sqrt(pow(b, 2) - 4 * a * c)) / (2 * a);
	ret->in = sphere;
	ret->out = sphere;
	ret->inPosition = d1 < d2 ? SumPoints(MulPoints(l, d1), camera) : SumPoints(MulPoints(l, d2), camera);
	ret->outPosition = d1 < d2 ? SumPoints(MulPoints(l, d2), camera) : SumPoints(MulPoints(l, d1), camera);
	return ret;
}
void DrawElement(Node* node, unsigned char *data, Position* camera) {
	if (node->left != NULL) {
		DrawElement(node->left, data, camera);
	}
	if (node->right != NULL) {
		DrawElement(node->right, data, camera);
	}
	for (int i = 0; i < LEN; i++) {
		for (int j = 0; j < LEN; j++) {
			Position* place = new Position();
			if (node->sphere != NULL) {
				Line* sphereLine = countLine(node->sphere, camera);
				if (sphereLine != NULL) {
					node->lines.push_back(sphereLine);
				}
				else {
					node->left->lines.insert(node->left->lines.end(), node->right->lines.begin(), node->right->lines.end());
					node->lines = node->left->lines;
				}
			}
		}
	}
	if (node->parent == NULL) {
		for (int i = 0; i < LEN; i++) {
			for (int j = 0; j < LEN; j++) {
				setInd(data, i, j no)
			}
		}
	}
	glDrawPixels(LEN, LEN, GL_RGBA, GL_UNSIGNED_BYTE, data);
}
#pragma endregion
#pragma region THREE


Sphere* setSpherePosition(Sphere* sphere, int x, int y, int z) {
	sphere->position.x = x;
	sphere->position.y = y;
	sphere->position.z = z;
	return sphere;
}
Sphere* setSphereColor(Sphere* sphere, int r, int g, int b, int a) {
	sphere->color.r = r;
	sphere->color.g = g;
	sphere->color.b = b;
	sphere->color.a = a;
	return sphere;
}
void CreateRoot() {
	root = new Node();
	root->operation = Sum;

	Sphere* sphere1 = new Sphere();
	sphere1->r = 200;
	sphere1 = setSpherePosition(sphere1, 100, 100, 0);
	sphere1 = setSphereColor(sphere1, 255, 0, 255, 255);

	Sphere* sphere3 = new Sphere();
	sphere3->r = 100;
	sphere3 = setSpherePosition(sphere3, -50, -50, 0);
	sphere3 = setSphereColor(sphere3, 0, 255, 0, 255);

	Sphere* sphere2 = new Sphere();
	sphere2->r = 100;
	sphere2 = setSpherePosition(sphere2, -120, -120, 0);
	sphere2 = setSphereColor(sphere2, 0, 0, 255, 255);

	Sphere* sphere4 = new Sphere();
	sphere4->r = 50;
	sphere4 = setSpherePosition(sphere4, 50, -50, 0);
	sphere4 = setSphereColor(sphere4, 100, 100, 100, 255);

	Node* left1 = new Node();
	left1->operation = Diff;
	left1->parent = root;

	Node* right2 = new Node();
	right2->operation = None;
	right2->sphere = sphere2;
	right2->parent = left1;

	Node* left2 = new Node();
	left2->operation = Sum;
	left2->parent = left1;

	left1->right = right2;
	left1->left = left2;

	Node* right3 = new Node();
	right3->operation = None;
	right3->sphere = sphere4;
	right3->parent = left2;

	Node* left3 = new Node();
	left3->operation = None;
	left3->sphere = sphere3;
	left3->parent = left2;

	left2->left = left3;
	left2->right = right3;

	Node* right = new Node();
	right->sphere = sphere1;
	right->operation = None;
	right->parent = root;
	root->right = right;
	root->left = left1;
}
#pragma endregion
#pragma region Render
void renderCpu() {
}
void renderGpu()
{
}
void render()
{
	Position* camera = new Position();
	camera->x = 0;
	camera->y = 0;
	camera->z = 200;
	glClearColor(0.0 / 255.0, 0.0 / 255.0, 0.0 / 225.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	DrawElement(root, prepareData(), camera);


	renderGpu();
	renderCpu();
	glutSwapBuffers();
	//glutPostRedisplay();
}
#pragma endregion
int main(int argc, char* argv[])
{
	// Initialize GLUTx
	glutInit(&argc, argv);
	// Set up some memory buffers for our display
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	// Set the window size
	glutInitWindowSize(LEN, LEN);
	// Create the window with the title "Hello,GL"
	glutCreateWindow("CSGThree");
	CreateRoot();
	glutDisplayFunc(render);
	glutMainLoop();

	GLenum err = glewInit();
	if (GLEW_OK != err) {
		fprintf(stderr, "GLEW error");
		return 1;
	}

	return 0;
}